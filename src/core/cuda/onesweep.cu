#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath> 

#define RADIX_BITS 8
#define RADIX 256
__global__ void globalBinCounter(int* input_array, int* global_counter, int numPasses, int N){
    extern __shared__ int local_counter[]; //warps * RADIX * numPasses
    int warpId = threadIdx.x >> 5;
    int laneId = threadIdx.x & 31;
    int numWarps = 8;
    int *start = local_counter + (warpId * (RADIX) * numPasses);

    for (int j = laneId; j < RADIX*numPasses; j += 32) {
        start[j] = 0;
    }
    __syncthreads();

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += gridDim.x * blockDim.x) {
        int v = input_array[i];
        #pragma unroll
        for (int p = 0; p < 4; ++p) {

            unsigned d = (v >> (p * RADIX_BITS)) & (RADIX - 1);
            atomicAdd(&start[p * RADIX + d], 1u);
            
        }
    }
    __syncthreads();

    for (int i = threadIdx.x; i < numWarps*numPasses; i += blockDim.x) {
        int sum = 0;
        for (int j = 0; j < RADIX; j++) {
            int idx = i * RADIX + j;
            int c = local_counter[idx];
            local_counter[idx] = sum;
            sum += c;
        }
    }
    __syncthreads();



    for (int j = threadIdx.x; j < RADIX * numPasses; j += blockDim.x) {
        unsigned int sum = 0u;
        #pragma unroll
        for (int w = 0; w < numWarps; ++w) sum += local_counter[w * RADIX * numPasses + j];
        atomicAdd(&global_counter[j], sum);
    }

}
__device__ int gTileCounter;
#define TILE_SIZE 1024
__global__ void oneSweep(int* input_array, int* output_array, int* lookback, int* global_counter, int N, int shift) {
    __shared__ int shared_current_tile;
    if (threadIdx.x == 0) shared_current_tile = atomicAdd(&gTileCounter, 1);
    __syncthreads();
    int current_tile = shared_current_tile;
    const int numTiles   = (N + TILE_SIZE - 1) / TILE_SIZE;
    if (current_tile >= numTiles) return;
    __shared__ int local_offset[TILE_SIZE];
    __shared__ int counter_full[RADIX*(9)]; //( num warps + 1)* RADIX
    __shared__ int tile_offset[RADIX];

    

    int warpId = threadIdx.x >> 5;
    int laneId = threadIdx.x & 31;
    int numWarps = 8;

    int chunk = TILE_SIZE / 256;
    int* counter = counter_full;

    int start = current_tile * TILE_SIZE + warpId * 32 * chunk + laneId;
    int end = min(N, current_tile * TILE_SIZE + (warpId+1) * 32 * chunk);

    for (int j = threadIdx.x; j < RADIX * (numWarps + 1); j += blockDim.x) counter_full[j] = 0;
    __syncthreads();

    for (int i = laneId + warpId * 32; i < TILE_SIZE; i += 256) {
        local_offset[i] = 0;
    }
    __syncthreads();
    unsigned mask[8];
    int pop;
    int offset = 0;
    for (int i = start; i < end; i += 32) {
        unsigned active = __activemask();
        int v = (input_array[i] >> (shift*RADIX_BITS)) & (RADIX - 1);
        //grab ballots from all threads in the warp
        for (int j = 0; j < 8; ++j) {
            mask[j] = __ballot_sync(active, (v >> j) & 1);
        }
        //find the threads that have same digit as me
        unsigned my_group = active;
        for (int j = 0; j < 8; ++j) {
            //i tried not to introduce any if else to prevent warp divergence
            unsigned bit   = (v >> j) & 1u;
            unsigned keep  = bit ? mask[j] : (active ^ mask[j]);  // complement within 'active'
            my_group      &= keep;
        }
        pop = __popc(my_group);
        int leader = __ffs(my_group) - 1;
        if (laneId == leader) {
            offset = atomicAdd(&counter[warpId * RADIX + v], pop);
        }
        offset = __shfl_sync(my_group, offset, leader);

        local_offset[i - current_tile * TILE_SIZE] = offset + __popc(my_group & ((1u << laneId) - 1));
    }
    __syncthreads();
    if (threadIdx.x < RADIX) {
        int sum = 0;
        #pragma unroll
        for (int w = 0; w < numWarps; ++w) {
            int c = counter[w * RADIX + threadIdx.x];   // this warp’s count
            counter[w * RADIX + threadIdx.x] = sum;           // exclusive prefix for this warp
            sum += c;
        }
        // store this tile’s *local* count per digit (for chaining)
        counter_full[numWarps * RADIX + threadIdx.x] = sum;   // tile_count[d]
    }
    __syncthreads();    
    
    // // if (j >= 0) counter_full[8*RADIX + j] += counter_full[7*RADIX + j];
    // // __syncthreads();
    unsigned flag;
    if (current_tile == 0) {
        flag = (1<<31); //first tile
    } else {
        flag = (1<<30);
    }

    for (int k = threadIdx.x; k < RADIX; k += blockDim.x) {
        atomicExch(&lookback[current_tile * (RADIX) + k], counter_full[8*RADIX + k] | flag);
        // lookback[current_tile * (RADIX) + k] = counter_full[8*RADIX + k] | flag;
    }
    __syncthreads();

    for (int k = threadIdx.x; k < RADIX; k += blockDim.x) {
            tile_offset[k] = 0;
    }
    __syncthreads();

    if (current_tile > 0){

        for (int k = threadIdx.x; k < RADIX; k += blockDim.x) {
            int prev_tile = (current_tile - 1) * RADIX;
            while (prev_tile >= 0){
                int flag = atomicAdd(&lookback[prev_tile + k], 0);
                bool global = ((flag >> 31) & 1) != 0;
                bool local  = ((flag >> 30) & 1) != 0;

                if (global){
                    tile_offset[k] += (flag & 0x3FFFFFFF);
                    break;
                } else if (local){
                    tile_offset[k] += (flag & 0x3FFFFFFF);
                    prev_tile -= RADIX;
                }else {
                    __nanosleep(64);
                }
            }

            // lookback[current_tile*(RADIX) + k] = (counter_full[8*RADIX + k] + tile_offset[k]) | (1 << 31);
            atomicExch(&lookback[current_tile*(RADIX) + k], (counter_full[8*RADIX + k] + tile_offset[k]) | (1 << 31));
        }
        __syncthreads();
    }

    
    __syncthreads();
    //start to scatter globally

    for (int i = start; i < end; i += 32) {
        int v = (input_array[i] >> (shift*RADIX_BITS)) & (RADIX - 1);
        int global_offset = global_counter[shift * RADIX + v];
        int pos = global_offset + tile_offset[v] + counter_full[warpId * RADIX + v] + local_offset[i - current_tile * TILE_SIZE];
        output_array[pos] = input_array[i];
    }

}

extern "C" void oneSweepSort(int* input, int* output, int N, int maxVal, float* kernel_ms){
    int* d_global_counter;
    int* d_lookback;
    int* d_input;
    int* d_output;
    hipMalloc(&d_output, N * sizeof(int));
    int numPasses = 4;

    int BLOCK_SIZE = 256;
    int NUM_BLOCKS = 174;

    int zero = 0;

    int TOTAL_TILES = (N + TILE_SIZE - 1) / TILE_SIZE;
    int SHARED_MEMORY_SIZE = RADIX * numPasses * (BLOCK_SIZE/32) * sizeof(int);

    hipMalloc(&d_global_counter, numPasses * (RADIX ) * sizeof(int));
    hipMalloc(&d_lookback, TOTAL_TILES * (RADIX) * sizeof(int));
    hipMalloc(&d_input, N * sizeof(int));
    hipMemset(d_global_counter, 0, numPasses * (RADIX) * sizeof(int));

    memcpy(output, input, N * sizeof(int));
    hipMemcpy(d_input, input, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_output, input, N * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t ev_start, ev_stop;
    hipEventCreate(&ev_start);
    hipEventCreate(&ev_stop);

    hipEventRecord(ev_start);

    globalBinCounter<<<NUM_BLOCKS, BLOCK_SIZE, SHARED_MEMORY_SIZE>>>(d_input, d_global_counter, numPasses, N);
    for (int shift = 0; shift < numPasses; ++shift) {
        hipMemcpyToSymbol(HIP_SYMBOL(gTileCounter), &zero, sizeof(int));
        hipMemset(d_lookback, 0, TOTAL_TILES * (RADIX) * sizeof(int));
        oneSweep<<<TOTAL_TILES, BLOCK_SIZE>>>(d_input, d_output, d_lookback, d_global_counter, N, shift);
        hipMemcpy(d_input, d_output, N * sizeof(int), hipMemcpyDeviceToDevice);
    }

    hipEventRecord(ev_stop);
    hipEventSynchronize(ev_stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, ev_start, ev_stop);
    *kernel_ms = milliseconds;

    hipEventDestroy(ev_start);
    hipEventDestroy(ev_stop);
    hipMemcpy(output, d_output, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_global_counter);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_lookback);
}