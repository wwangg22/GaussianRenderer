#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gaussians.hpp"
#include "render.cuh"

static __global__ void globalBinCounter(lightWeightGaussian* d_in, int* d_global_counter, int numPasses, int N) {
    extern __shared__ int local_counter[]; //warps * RADIX * numPasses
    int warpId = threadIdx.x >> 5;
    int laneId = threadIdx.x & 31;
    int numWarps = 8;
    int *start = local_counter + (warpId * (RADIX) * numPasses);

    for (int j = laneId; j < RADIX*numPasses; j += 32) {
        start[j] = 0;
    }
    __syncthreads();

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += gridDim.x * blockDim.x) {
        uint64_t v = d_in[i].radix_id;
        for (int p = 0; p < numPasses; ++p) {
            unsigned d = (v >> (p * RADIX_BITS)) & (RADIX - 1);
            atomicAdd(&start[p * RADIX + d], 1u);
        }
    }
    __syncthreads();

    for (int i = threadIdx.x; i < numWarps*numPasses; i += blockDim.x) {
        int sum = 0;
        for (int j = 0; j < RADIX; j++) {
            int idx = i * RADIX + j;
            int c = local_counter[idx];
            local_counter[idx] = sum;
            sum += c;
        }
    }
    __syncthreads();

    for (int j = threadIdx.x; j < RADIX * numPasses; j += blockDim.x) {
        unsigned int sum = 0u;
        #pragma unroll
        for (int w = 0; w < numWarps; ++w) sum += local_counter[w * RADIX * numPasses + j];
        atomicAdd(&d_global_counter[j], sum);
    }
}

static __global__ void oneSweep(lightWeightGaussian* d_in, lightWeightGaussian* d_out, int* lookback, int* global_counter, int N, int shift) {
    // __shared__ int shared_current_tile;
    // if (threadIdx.x == 0) shared_current_tile = atomicAdd(&gTileCounter, 1);
    // __syncthreads();
    // int current_tile = shared_current_tile;
    int current_tile = blockIdx.x;
    const int numTiles   = (N + TILE_SIZE - 1) / TILE_SIZE;
    if (current_tile >= numTiles) return;
    // __shared__ int local_offset[TILE_SIZE];
    // __shared__ int local_value[TILE_SIZE];
    __shared__ int counter_full[RADIX*(9)]; //( num warps + 1)* RADIX
    __shared__ int tile_offset[RADIX];


    int warpId = threadIdx.x >> 5;
    int laneId = threadIdx.x & 31;
    int numWarps = blockDim.x / 32;

    int chunk = TILE_SIZE / 256;
    int local_offset[8]; //TILE_SIZE  / 256
    lightWeightGaussian local_value[8]; //  TILE_SIZE  / 256
    int* counter = counter_full;

    int start = current_tile * TILE_SIZE + warpId * 32 * chunk + laneId;
    int end =  current_tile * TILE_SIZE + (warpId+1) * 32 * chunk;

    for (int j = threadIdx.x; j < RADIX * (numWarps + 1); j += blockDim.x) counter_full[j] = 0;
    __syncthreads();

    // for (int i = laneId + warpId * 32; i < TILE_SIZE; i += blockDim.x) {
    //     local_offset[i] = 0;
    // }
    #pragma unroll
    for (int i =0; i < 4; ++i){
        local_offset[i] = 0;
    }
    __syncthreads();
    unsigned mask[RADIX_BITS];
    int pop;
    int offset = 0;
    int part;
    for (int i = start; i < end; i += 32) {
        if (i >= N) part  = 0;
        else part = 1;
        unsigned active = __ballot_sync(0xFFFFFFFF, part);
        if (!part) continue;
        lightWeightGaussian lwg = d_in[i];
        uint64_t val = lwg.radix_id;
        // local_value[i - current_tile * TILE_SIZE] = val;
        local_value[(i - start) / 32] = lwg;
        int v = (val >> (shift*RADIX_BITS)) & (RADIX - 1);

        unsigned my_group = __match_any_sync(active,v);
        pop = __popc(my_group);
        int leader = __ffs(my_group) - 1;
        if (laneId == leader) {
            offset = counter[warpId * RADIX + v];
            counter[warpId * RADIX + v] += pop;
        }
        offset = __shfl_sync(my_group, offset, leader);

        // local_offset[i - current_tile * TILE_SIZE] = offset + __popc(my_group & ((1u << laneId) - 1));
        local_offset[(i - start) / 32] = offset + __popc(my_group & ((1u << laneId) - 1));
    }
    __syncthreads();
    if (threadIdx.x < RADIX) {
        int sum = 0;
        for (int w = 0; w < numWarps; ++w) {
            int c = counter[w * RADIX + threadIdx.x];   // this warp’s count
            counter[w * RADIX + threadIdx.x] = sum;           // exclusive prefix for this warp
            sum += c;
        }
        // store this tile’s *local* count per digit (for chaining)
        counter_full[numWarps * RADIX + threadIdx.x] = sum;   // tile_count[d]
    }
    __syncthreads();    
    
    // // if (j >= 0) counter_full[8*RADIX + j] += counter_full[7*RADIX + j];
    // // __syncthreads();
    unsigned flag;
    if (current_tile == 0) {
        flag = (1<<31); //first tile
    } else {
        flag = (1<<30);
    }

    for (int k = threadIdx.x; k < RADIX; k += blockDim.x) {
        __threadfence();
        // atomicExch(&lookback[current_tile * (RADIX) + k], counter_full[8*RADIX + k] | flag);
        lookback[current_tile * (RADIX) + k] = counter_full[8*RADIX + k] | flag;
        // lookback[current_tile * (RADIX) + k] = counter_full[8*RADIX + k] | flag;
    }
    __syncthreads();

    for (int k = threadIdx.x; k < RADIX; k += blockDim.x) {
            tile_offset[k] = 0;
    }
    __syncthreads();

    if (current_tile > 0){

        for (int k = threadIdx.x; k < RADIX; k += blockDim.x) {
            int prev_tile = (current_tile - 1) * RADIX;
            while (prev_tile >= 0){
                // int flag = atomicAdd(&lookback[prev_tile + k], 0);
                __threadfence();
                int flag = lookback[prev_tile + k];
                bool global = ((flag >> 31) & 1) != 0;
                bool local  = ((flag >> 30) & 1) != 0;

                if (global){
                    tile_offset[k] += (flag & 0x3FFFFFFF);
                    break;
                } else if (local){
                    tile_offset[k] += (flag & 0x3FFFFFFF);
                    prev_tile -= RADIX;
                }else {
                }
            }
            __threadfence();
            lookback[current_tile*(RADIX) + k] = (counter_full[8*RADIX + k] + tile_offset[k]) | (1 << 31);
            // atomicExch(&lookback[current_tile*(RADIX) + k], (counter_full[8*RADIX + k] + tile_offset[k]) | (1 << 31));
        }
        __syncthreads();
    }
    
    for (int k = threadIdx.x; k < RADIX; k += blockDim.x) {
        tile_offset[k] += global_counter[shift * RADIX + k];
    }
        //start to scatter globally
    __syncthreads();

    for (int i = start; i < end; i += 32) {
        if (i >= N) continue;
        // int v = (local_value[i - current_tile * TILE_SIZE] >> (shift*RADIX_BITS)) & (RADIX - 1);
        int v = (local_value[(i - start) / 32].radix_id >> (shift*RADIX_BITS)) & (RADIX - 1);
        // int pos = tile_offset[v] + counter_full[warpId * RADIX + v] + local_offset[i - current_tile * TILE_SIZE];
        // output_array[pos] = local_value[i - current_tile * TILE_SIZE];
        int pos = tile_offset[v] + counter_full[warpId * RADIX + v] + local_offset[(i - start) / 32];
        d_out[pos] = local_value[(i - start) / 32];
    }

}

extern "C" void oneSweep3DGaussianSort(lightWeightGaussian* d_in, 
                                       int N, 
                                       int num_bits,
                                       float* kernel_ms) {
    
    int* d_global_counter;
    int* d_lookback;
    lightWeightGaussian* d_input;
    lightWeightGaussian* d_output;
    hipMalloc(&d_output, N * sizeof(lightWeightGaussian));
    int numPasses = (num_bits + 7) / 8;

    int BLOCK_SIZE = 256;
    int NUM_BLOCKS = 174;

    int zero = 0;

    int TOTAL_TILES = (N + TILE_SIZE - 1) / TILE_SIZE;
    int SHARED_MEMORY_SIZE = RADIX * numPasses * (BLOCK_SIZE/32) * sizeof(int);

    hipMalloc(&d_global_counter, numPasses * (RADIX ) * sizeof(int));
    hipMalloc(&d_lookback, TOTAL_TILES * (RADIX) * sizeof(int));
    hipMalloc(&d_input, N * sizeof(lightWeightGaussian));
    hipMemset(d_global_counter, 0, numPasses * (RADIX) * sizeof(int));

    hipMemcpy(d_input, d_in, N * sizeof(lightWeightGaussian), hipMemcpyHostToDevice);
    hipMemcpy(d_output, d_in, N * sizeof(lightWeightGaussian), hipMemcpyHostToDevice);
    hipEvent_t ev_start, ev_stop;
    hipEventCreate(&ev_start);
    hipEventCreate(&ev_stop);

    hipEventRecord(ev_start);
    lightWeightGaussian* in = d_input;
    lightWeightGaussian* out = d_output;

    globalBinCounter<<<NUM_BLOCKS, BLOCK_SIZE, SHARED_MEMORY_SIZE>>>(d_input, d_global_counter, numPasses, N);

    for (int shift = 0; shift < numPasses; ++shift) {
        // hipMemcpyToSymbol(HIP_SYMBOL(gTileCounter), &zero, sizeof(int));
        hipMemset(d_lookback, 0, TOTAL_TILES * (RADIX) * sizeof(int));
        oneSweep<<<TOTAL_TILES, BLOCK_SIZE>>>(in, out, d_lookback, d_global_counter, N, shift);
        std::swap(in, out);
        // hipMemcpy(d_input, d_output, N * sizeof(int), hipMemcpyDeviceToDevice);
    }

    hipEventRecord(ev_stop);
    hipEventSynchronize(ev_stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, ev_start, ev_stop);
    *kernel_ms = milliseconds;

    hipEventDestroy(ev_start);
    hipEventDestroy(ev_stop);
    lightWeightGaussian* result = (numPasses & 1) ? in : out; 
    hipMemcpy(d_in, result, N * sizeof(lightWeightGaussian), hipMemcpyDeviceToHost);

    hipFree(d_global_counter);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_lookback);
}

static __global__ void renderGaussians(float* out_pixels, TilingInformation* tile_info, Gaussian* gaussians, lightWeightGaussian* sorted_gaussians ) {
    extern __shared__ float buf[];
    float* shared_T = buf; // tile_info->height_stride * tile_info->width_stride
    float* shared_rgb = buf + tile_info->height_stride * tile_info->width_stride; // 3 * tile_info->height_stride * tile_info->width_stride
    int block_size = tile_info->height_stride * tile_info->width_stride;
    __shared__ Gaussian cur_gauss;
    __shared__ lightWeightGaussian cur_lwg;
    for (int grid_id = blockIdx.x; grid_id < tile_info->num_tile_y * tile_info->num_tile_x; grid_id += gridDim.x) {

        int tile_x = grid_id % tile_info->num_tile_x;
        int tile_y = grid_id / tile_info->num_tile_x;

        int x_offset = tile_x * tile_info->width_stride;
        int y_offset = tile_y * tile_info->height_stride;

        int tile_offset_end = tile_info->tile_id_offset[grid_id];
        int tile_offset_start;

        if (grid_id == 0) {
            tile_offset_start = 0;
        } else {
            tile_offset_start = tile_info->tile_id_offset[grid_id - 1];
        }

        for (int idx = tile_offset_start; idx < tile_offset_end; ++idx) {
            if (threadIdx.x == 0) {
                cur_lwg = sorted_gaussians[idx];
                cur_gauss = gaussians[cur_lwg.gaussian_id];
            }
            __syncthreads();

            int px_x = cur_gauss.px_x;
            int px_y = cur_gauss.px_y;

            int aabb_xmin = cur_gauss.aabb[0];
            int aabb_ymin = cur_gauss.aabb[1];
            int aabb_xmax = cur_gauss.aabb[2];
            int aabb_ymax = cur_gauss.aabb[3];

            float inv_covar[4];
            inv_covar[0] = cur_gauss.inv_covar[0];
            inv_covar[1] = cur_gauss.inv_covar[1];
            inv_covar[2] = cur_gauss.inv_covar[2];
            inv_covar[3] = cur_gauss.inv_covar[3];

            for (int j = threadIdx.x; j < tile_info->height_stride * tile_info->width_stride; j += blockDim.x) {
                int global_x = j % tile_info->width_stride + x_offset;
                int global_y = j / tile_info->width_stride + y_offset;
                if (global_x >= tile_info->W || global_y >= tile_info->H) continue;
                if (global_x < aabb_xmin || global_x > aabb_xmax || global_y < aabb_ymin || global_y > aabb_ymax) continue;
                if (shared_T[j] < 1e-3f) continue;

                float dx = (static_cast<float>(global_x) - static_cast<float>(px_x));
                float dy = (static_cast<float>(global_y) - static_cast<float>(px_y));

                float md2 = dx * (inv_covar[0]*dx + inv_covar[1]*dy) + dy * (inv_covar[2]*dx + inv_covar[3]*dy);
                float opacity = cur_gauss.opacity * expf(-0.5f * md2);

                for (int c = 0; c < 3; ++c) {
                    shared_rgb[c*block_size + j] += cur_gauss.color[c] * opacity * shared_T[j];
                }
                shared_T[j] *= (1.0f - opacity);
            }

        }

        __syncthreads();

        for (int j = threadIdx.x; j < tile_info->height_stride * tile_info->width_stride; j += blockDim.x) {
            for (int c = 0; c< 3; c++) {
                out_pixels[c*(tile_info->H * tile_info->W) + (y_offset + j / tile_info->width_stride) * tile_info->W + (x_offset + j % tile_info->width_stride)] += shared_rgb[c*block_size + j];
                shared_rgb[c*block_size + j] = 0.0f;
            }
            shared_T[j] = 1.0f;
        }
        __syncthreads();
        
    }
}

extern "C" void renderGaussiansCUDA(float* d_out_pixels, 
                                 TilingInformation* d_tile_info, 
                                 Gaussian* d_gaussians, 
                                 lightWeightGaussian* d_sorted_gaussians,
                                 int num_gaussians,
                                 int num_lwg,
                                 float* kernel_ms) {
    hipEvent_t ev_start, ev_stop;
    hipEventCreate(&ev_start);
    hipEventCreate(&ev_stop);

    hipEventRecord(ev_start);
    float* d_out;
    TilingInformation* d_ti;
    Gaussian* d_g;
    lightWeightGaussian* d_sg;


    hipMalloc(&d_out, sizeof(float) * 3 * d_tile_info->H * d_tile_info->W);
    hipMalloc(&d_ti, sizeof(TilingInformation));
    hipMalloc(&d_g, sizeof(Gaussian) * num_gaussians);
    hipMalloc(&d_sg, sizeof(lightWeightGaussian) * num_lwg);

    int BLOCK_SIZE = 256;
    int NUM_BLOCKS = 128;

    size_t shared_mem_size = (d_tile_info->height_stride * d_tile_info->width_stride) * (1 + 3) * sizeof(float);

    renderGaussians<<<NUM_BLOCKS, BLOCK_SIZE, shared_mem_size>>>(d_out, d_ti, d_g, d_sg);

    hipEventRecord(ev_stop);
    hipEventSynchronize(ev_stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, ev_start, ev_stop);
    *kernel_ms = milliseconds;

    hipEventDestroy(ev_start);
    hipEventDestroy(ev_stop);
}